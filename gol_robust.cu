#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <climits>
#include <algorithm>

void readField(std::vector<uint8_t> &field, int height) {
  std::string inputLine;
  for (int i = 0; i <= height; i++)
  {
    std::getline(std::cin, inputLine);
    std::vector<uint8_t> tempvec;
    std::istringstream input(inputLine);
    std::string temp;

    while (std::getline(input, temp, ' '))
    {
      if (temp == "X")
        tempvec.emplace_back(1);
      else if (temp == "O")
        tempvec.emplace_back(0);
      else
        std::cerr << "symbol mismatch\n";
    }
    for (uint8_t i : tempvec) {
      field.emplace_back(i);
    }
  }
}

void writeField(std::vector<uint8_t> &v, int width) {
  int count = 0;
  for (uint8_t i : v)
  {
    count++;
    // std::cout << static_cast<int>(i) << " ";
    std::cout << ((i == 0) ? 'O' : 'X') << " ";
    if (count % width == 0)
      std::cout << std::endl;
  }
}

hipError_t lifeWithCuda(uint8_t *field, unsigned int width, unsigned int height, unsigned int generation);

__global__ void lifeKernelaggregator(uint8_t *field, uint8_t *tempfield, unsigned int width, unsigned int height)
{
  for (int b = blockIdx.x; b < height; b += gridDim.x) { // rows are always exclusive to blocks, they can only grow to height, thus never out of the size of the memory segment
    for (int t = threadIdx.x; t < width; t += blockDim.x) // threads not growing past the size of width ensures threads not accessing memory where they shouldn't
    {
      // establish identity of cell 
      int cellnr = b * width + t;
      // calculate 1d aggregated neigbourhood (top + mid + bot) and drop in temp
      if (b == 0)
        tempfield[cellnr] = field[(width * (height - 1)) + t];
      else
        tempfield[cellnr] = field[cellnr - width];  // row on top

      tempfield[cellnr] += field[cellnr];

      if (b == (height - 1))
        tempfield[cellnr] += field[(0 * width) + t];
      else
        tempfield[cellnr] += field[cellnr + width]; //row below
      __syncthreads(); // obsolete?
    }
  }
}

__global__ void lifeKernel(uint8_t *field, uint8_t *tempfield, unsigned int width, unsigned int height)
{
  for (int b = blockIdx.x; b < height; b += gridDim.x) { // rows are always exclusive to blocks, they can only grow to height, thus never out of the size of the memory segment
    for (int t = threadIdx.x; t < width; t += blockDim.x) // threads not growing past the size of width ensures threads not accessing memory where they shouldn't
    {
      int cellnr = b * width + t;
      // calculate cell value
      uint8_t left;
      if (t == 0)
        left = tempfield[cellnr + width - 1];
      else
        left = tempfield[cellnr - 1];
      uint8_t mid = tempfield[cellnr];
      uint8_t right;
      if (t == width - 1)
        right = tempfield[cellnr + 1 - width];
      else
        right = tempfield[cellnr + 1];

      if (field[cellnr] == 1) {
        if (!(3 <= left + mid + right && left + mid + right <= 4))
          field[cellnr] = 0;
      }
      else
      {
        if (left + mid + right == 3)
          field[cellnr] = 1;
      }
      __syncthreads();
    }
  }
}

int main()
{
  int generations;
  std::cin >> generations;
  int width;
  std::cin >> width;
  int height;
  std::cin >> height;
  std::vector<uint8_t> field; //( width, std::vector<float> ( height, 0 ) ) //could initialize

  readField(field, height);

  // generate generation X of game of life on field
  hipError_t cudaStatus = lifeWithCuda(field.data(), width, height, generations);
  if (cudaStatus != hipSuccess) {
    std::cerr << "lifeWithCuda failed!\n";
    return 1;
  }

  writeField(field, width);

  // cudaDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Nsight and Visual Profiler to show complete traces.
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipDeviceReset failed!\n";
    return 1;
  }

  return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t lifeWithCuda(uint8_t *field, unsigned int width, unsigned int height, unsigned int generation)
{
  uint8_t *dev_field = 0;
  uint8_t *dev_field_temp = 0;
  hipError_t cudaStatus;

  
  // for robustness against enormous entries
  unsigned int maxThreadsperBlock = 1024; // for my device maxThreadsperBlock is 1024
  unsigned int maxBlocksperGrid = 12288;   // max gridx is 2147483647, but shared memory per block is just 49152 Bytes, i assume 3 times my entry as memory usage, so i pick ~1/4th to be safe
  int threadnum = std::min(maxThreadsperBlock, width); 
  int blocknum = std::min(maxBlocksperGrid, height); 

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n";
    goto Error;
  }

  // Allocate GPU buffers for two fields (one input, one output)    .
  cudaStatus = hipMalloc((void**)&dev_field, height * (width * sizeof(uint8_t)));
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMalloc failed!\n";
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&dev_field_temp, height * (width * sizeof(uint8_t)));
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMalloc failed!\n";
    goto Error;
  }

  // Copy input field from host memory to GPU buffers.
  cudaStatus = hipMemcpy(dev_field, field, height * (width * sizeof(uint8_t)), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMemcpy failed!\n";
    goto Error;
  }

  // TODO: figure aout, what kind of thread distribution works best
  for (unsigned int g = 0; g < generation; g++)
  {
    lifeKernelaggregator <<<blocknum, threadnum >>> (dev_field, dev_field_temp, width, height);
    lifeKernel <<<blocknum, threadnum >>> (dev_field, dev_field_temp, width, height);
    std::cout << ".";
  }
  std::cout << std::endl;

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    std::cerr << "addKernel launch failed: %s\n" << hipGetErrorString(cudaStatus);
    goto Error;
  }

  // cudaDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipDeviceSynchronize returned error code %d after launching addKernel!\n" << cudaStatus;
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(field, dev_field, height * (width * sizeof(uint8_t)), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMemcpy failed!\n";
    goto Error;
  }

Error:
  hipFree(dev_field);
  hipFree(dev_field_temp);

  return cudaStatus;
}
